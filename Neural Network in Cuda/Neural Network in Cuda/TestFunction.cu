#include "hip/hip_runtime.h"
#include "TestFunction.cuh"

__global__ void SetIntArray(int* data, int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size)
	{
		data[index] = index;
	}
}

TestFunctionAPI int TestFunction()
{
	return 6666;
}

TestFunctionAPI int* TestCudaFunction(int size)
{
	int ThreadSize = 256;
	int BlockSize = size / ThreadSize + 1;

	int *HostDataArray = new int[size];
	int *DeviceDataArray;

	size_t DataSize = sizeof(int) * size;
	hipMalloc(&DeviceDataArray, DataSize);
	SetIntArray << <BlockSize, ThreadSize >> > (DeviceDataArray, size);

	hipMemcpy(HostDataArray, DeviceDataArray, DataSize, hipMemcpyDeviceToHost);
	hipFree(DeviceDataArray);

	return HostDataArray;
}

TestFunctionAPI void TestFreeFunction(int *data)
{
	delete[] data;
}

